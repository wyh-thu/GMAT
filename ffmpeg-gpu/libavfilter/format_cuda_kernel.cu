#include "hip/hip_runtime.h"
/*
* Copyright 2017-2020 NVIDIA Corporation.  All rights reserved.
*
* Please refer to the NVIDIA end user license agreement (EULA) associated
* with this source code for terms and conditions that govern your use of
* this software. Any use, reproduction, disclosure, or distribution of
* this software and related documentation outside the terms of the EULA
* is strictly prohibited.
*
*/

#include <cstddef>
#include <cstdint>
#include <stdint.h>
#include <stdio.h>
#ifdef __cplusplus
extern "C"
{
#endif
#include <hip/hip_runtime.h>

#include "libavutil/pixfmt.h"
#ifdef __cplusplus
}
#endif
#include "format_cuda.h"

__constant__ float matYuv2Rgb[3][3];
__constant__ float matRgb2Yuv[3][3];


void inline GetConstants(int iMatrix, float &wr, float &wb, int &black, int &white, int &max) {
    black = 16; white = 235;
    max = 255;

    switch (iMatrix)
    {
    case AVCOL_SPC_BT709:
    default:
        wr = 0.2126f; wb = 0.0722f;
        break;

    case AVCOL_SPC_FCC:
        wr = 0.30f; wb = 0.11f;
        break;

    case AVCOL_SPC_BT470BG:
        wr = 0.2990f; wb = 0.1140f;
        break;

    case AVCOL_SPC_SMPTE240M:
        wr = 0.212f; wb = 0.087f;
        break;

    case AVCOL_SPC_BT2020_NCL:
    case AVCOL_SPC_BT2020_CL:
        wr = 0.2627f; wb = 0.0593f;
        // 10-bit only
        black = 64 << 6; white = 940 << 6;
        max = (1 << 16) - 1;
        break;
    }
}

void SetMatYuv2Rgb(int iMatrix, hipStream_t stream = 0) {
    float wr, wb;
    int black, white, max;
    GetConstants(iMatrix, wr, wb, black, white, max);
    float mat[3][3] = {
        1.0f, 0.0f, (1.0f - wr) / 0.5f,
        1.0f, -wb * (1.0f - wb) / 0.5f / (1 - wb - wr), -wr * (1 - wr) / 0.5f / (1 - wb - wr),
        1.0f, (1.0f - wb) / 0.5f, 0.0f,
    };
    for (int i = 0; i < 3; i++) {
        for (int j = 0; j < 3; j++) {
            mat[i][j] = (float)(1.0 * max / (white - black) * mat[i][j]);
        }
    }
    hipMemcpyToSymbolAsync(HIP_SYMBOL(matYuv2Rgb), mat, sizeof(mat), 0, hipMemcpyHostToDevice, stream);
}

void SetMatRgb2Yuv(int iMatrix, hipStream_t stream = 0) {
    float wr, wb;
    int black, white, max;
    GetConstants(iMatrix, wr, wb, black, white, max);
    float mat[3][3] = {
        wr, 1.0f - wb - wr, wb,
        -0.5f * wr / (1.0f - wb), -0.5f * (1 - wb - wr) / (1.0f - wb), 0.5f,
        0.5f, -0.5f * (1.0f - wb - wr) / (1.0f - wr), -0.5f * wb / (1.0f - wr),
    };
    for (int i = 0; i < 3; i++) {
        for (int j = 0; j < 3; j++) {
            mat[i][j] = (float)(1.0 * (white - black) / max * mat[i][j]);
        }
    }
    hipMemcpyToSymbolAsync(HIP_SYMBOL(matRgb2Yuv), mat, sizeof(mat), 0, hipMemcpyHostToDevice, stream);
}

template<class T>
__device__ static T Clamp(T x, T lower, T upper) {
    return x < lower ? lower : (x > upper ? upper : x);
}

template<class Rgb, class YuvUnit>
__device__ static inline Rgb YuvToRgbForPixel(YuvUnit y, YuvUnit u, YuvUnit v) {
    const int
        low = 1 << (sizeof(YuvUnit) * 8 - 4),
        mid = 1 << (sizeof(YuvUnit) * 8 - 1);
    float fy = (int)y - low, fu = (int)u - mid, fv = (int)v - mid;
    const float maxf = (1 << sizeof(YuvUnit) * 8) - 1.0f;
    YuvUnit
        r = (YuvUnit)Clamp(matYuv2Rgb[0][0] * fy + matYuv2Rgb[0][1] * fu + matYuv2Rgb[0][2] * fv, 0.0f, maxf),
        g = (YuvUnit)Clamp(matYuv2Rgb[1][0] * fy + matYuv2Rgb[1][1] * fu + matYuv2Rgb[1][2] * fv, 0.0f, maxf),
        b = (YuvUnit)Clamp(matYuv2Rgb[2][0] * fy + matYuv2Rgb[2][1] * fu + matYuv2Rgb[2][2] * fv, 0.0f, maxf);

    Rgb rgb{};
    const int nShift = abs((int)sizeof(YuvUnit) - (int)sizeof(rgb.c.r)) * 8;
    if (sizeof(YuvUnit) >= sizeof(rgb.c.r)) {
        rgb.c.r = r >> nShift;
        rgb.c.g = g >> nShift;
        rgb.c.b = b >> nShift;
    } else {
        rgb.c.r = r << nShift;
        rgb.c.g = g << nShift;
        rgb.c.b = b << nShift;
    }
    return rgb;
}

template<>
__device__ inline RGBAF32 YuvToRgbForPixel(uint8_t y, uint8_t u, uint8_t v) {
    const int
        low = 1 << (sizeof(uint8_t) * 8 - 4),
        mid = 1 << (sizeof(uint8_t) * 8 - 1);
    float fy = (int)y - low, fu = (int)u - mid, fv = (int)v - mid;
    const float maxf = (1 << sizeof(uint8_t) * 8) - 1.0f;
    uint8_t
        r = (uint8_t)Clamp(matYuv2Rgb[0][0] * fy + matYuv2Rgb[0][1] * fu + matYuv2Rgb[0][2] * fv, 0.0f, maxf),
        g = (uint8_t)Clamp(matYuv2Rgb[1][0] * fy + matYuv2Rgb[1][1] * fu + matYuv2Rgb[1][2] * fv, 0.0f, maxf),
        b = (uint8_t)Clamp(matYuv2Rgb[2][0] * fy + matYuv2Rgb[2][1] * fu + matYuv2Rgb[2][2] * fv, 0.0f, maxf);

    RGBAF32 rgb{};

    rgb.c.r = r;
    rgb.c.g = g;
    rgb.c.b = b;
    return rgb;
}

template<>
__device__ inline BGRAF32 YuvToRgbForPixel(uint8_t y, uint8_t u, uint8_t v) {
    const int
        low = 1 << (sizeof(uint8_t) * 8 - 4),
        mid = 1 << (sizeof(uint8_t) * 8 - 1);
    float fy = (int)y - low, fu = (int)u - mid, fv = (int)v - mid;
    const float maxf = (1 << sizeof(uint8_t) * 8) - 1.0f;
    uint8_t
        r = (uint8_t)Clamp(matYuv2Rgb[0][0] * fy + matYuv2Rgb[0][1] * fu + matYuv2Rgb[0][2] * fv, 0.0f, maxf),
        g = (uint8_t)Clamp(matYuv2Rgb[1][0] * fy + matYuv2Rgb[1][1] * fu + matYuv2Rgb[1][2] * fv, 0.0f, maxf),
        b = (uint8_t)Clamp(matYuv2Rgb[2][0] * fy + matYuv2Rgb[2][1] * fu + matYuv2Rgb[2][2] * fv, 0.0f, maxf);

    BGRAF32 rgb{};

    rgb.c.r = r;
    rgb.c.g = g;
    rgb.c.b = b;
    return rgb;
}

template<class YuvUnitx2, class Rgb, class RgbIntx2>
__global__ static void YuvToRgbKernel(uint8_t *pYuv, int nYuvPitch, uint8_t *pRgb, int nRgbPitch, int nWidth, int nHeight) {
    int x = (threadIdx.x + blockIdx.x * blockDim.x) * 2;
    int y = (threadIdx.y + blockIdx.y * blockDim.y) * 2;
    if (x + 1 >= nWidth || y + 1 >= nHeight) {
        return;
    }

    uint8_t *pSrc = pYuv + x * sizeof(YuvUnitx2) / 2 + y * nYuvPitch;
    uint8_t *pDst = pRgb + x * sizeof(Rgb) + y * nRgbPitch;

    YuvUnitx2 l0 = *(YuvUnitx2 *)pSrc;
    YuvUnitx2 l1 = *(YuvUnitx2 *)(pSrc + nYuvPitch);
    YuvUnitx2 ch = *(YuvUnitx2 *)(pSrc + (nHeight - y / 2) * nYuvPitch);

    *(RgbIntx2 *)pDst = RgbIntx2 {
        YuvToRgbForPixel<Rgb>(l0.x, ch.x, ch.y).d,
        YuvToRgbForPixel<Rgb>(l0.y, ch.x, ch.y).d,
    };
    *(RgbIntx2 *)(pDst + nRgbPitch) = RgbIntx2 {
        YuvToRgbForPixel<Rgb>(l1.x, ch.x, ch.y).d,
        YuvToRgbForPixel<Rgb>(l1.y, ch.x, ch.y).d,
    };
}

template<class YuvUnitx2, class Rgb, class RgbIntx2>
__global__ static void Yuv444ToRgbKernel(uint8_t *pYuv, int nYuvPitch, uint8_t *pRgb, int nRgbPitch, int nWidth, int nHeight) {
    int x = (threadIdx.x + blockIdx.x * blockDim.x) * 2;
    int y = (threadIdx.y + blockIdx.y * blockDim.y);
    if (x + 1 >= nWidth || y  >= nHeight) {
        return;
    }

    uint8_t *pSrc = pYuv + x * sizeof(YuvUnitx2) / 2 + y * nYuvPitch;
    uint8_t *pDst = pRgb + x * sizeof(Rgb) + y * nRgbPitch;

    YuvUnitx2 l0 = *(YuvUnitx2 *)pSrc;
    YuvUnitx2 ch1 = *(YuvUnitx2 *)(pSrc + (nHeight * nYuvPitch));
    YuvUnitx2 ch2 = *(YuvUnitx2 *)(pSrc + (2 * nHeight * nYuvPitch));

    *(RgbIntx2 *)pDst = RgbIntx2{
        YuvToRgbForPixel<Rgb>(l0.x, ch1.x, ch2.x).d,
        YuvToRgbForPixel<Rgb>(l0.y, ch1.y, ch2.y).d,
    };
}

template<class YuvUnitx2, class Rgb, class RgbUnitx2>
__global__ static void YuvToRgbPlanarKernel(uint8_t *pYuv, int nYuvPitch, uint8_t *pRgbp, int nRgbpPitch, int nWidth, int nHeight) {
    int x = (threadIdx.x + blockIdx.x * blockDim.x) * 2;
    int y = (threadIdx.y + blockIdx.y * blockDim.y) * 2;
    if (x + 1 >= nWidth || y + 1 >= nHeight) {
        return;
    }

    uint8_t *pSrc = pYuv + x * sizeof(YuvUnitx2) / 2 + y * nYuvPitch;

    YuvUnitx2 l0 = *(YuvUnitx2 *)pSrc;
    YuvUnitx2 l1 = *(YuvUnitx2 *)(pSrc + nYuvPitch);
    YuvUnitx2 ch = *(YuvUnitx2 *)(pSrc + (nHeight - y / 2) * nYuvPitch);

    Rgb rgb0 = YuvToRgbForPixel<Rgb>(l0.x, ch.x, ch.y),
        rgb1 = YuvToRgbForPixel<Rgb>(l0.y, ch.x, ch.y),
        rgb2 = YuvToRgbForPixel<Rgb>(l1.x, ch.x, ch.y),
        rgb3 = YuvToRgbForPixel<Rgb>(l1.y, ch.x, ch.y);

    uint8_t *pDst = pRgbp + x * sizeof(RgbUnitx2) / 2 + y * nRgbpPitch;
    *(RgbUnitx2 *)pDst = RgbUnitx2 {rgb0.v.x, rgb1.v.x};
    *(RgbUnitx2 *)(pDst + nRgbpPitch) = RgbUnitx2 {rgb2.v.x, rgb3.v.x};
    pDst += nRgbpPitch * nHeight;
    *(RgbUnitx2 *)pDst = RgbUnitx2 {rgb0.v.y, rgb1.v.y};
    *(RgbUnitx2 *)(pDst + nRgbpPitch) = RgbUnitx2 {rgb2.v.y, rgb3.v.y};
    pDst += nRgbpPitch * nHeight;
    *(RgbUnitx2 *)pDst = RgbUnitx2 {rgb0.v.z, rgb1.v.z};
    *(RgbUnitx2 *)(pDst + nRgbpPitch) = RgbUnitx2 {rgb2.v.z, rgb3.v.z};
}

template<class Rgb>
__device__ static inline Rgb normalizePixel(Rgb pixel, float norm, float3 shift){
    Rgb rgb{};
    rgb.c.r = (pixel.c.r - shift.x) / norm;
    rgb.c.g = (pixel.c.g - shift.y) / norm;
    rgb.c.b = (pixel.c.b - shift.z) / norm;

    return rgb;
}

// Support shift and normalization
template<class YuvUnitx2, class Rgb, class RgbUnitx2>
__global__ static void YuvToRgbPlanarKernel(uint8_t *pY, uint8_t *pUV, int yLinesize, int uvLinesize,
                                            uint8_t *p1, uint8_t *p2, uint8_t *p3, int rgbpLinesize,
                                            int nWidth, int nHeight, float norm=255.0f, 
                                            float shift_r=0, float shift_g=0, float shift_b=0) {
    int x = (threadIdx.x + blockIdx.x * blockDim.x) * 2;
    int y = (threadIdx.y + blockIdx.y * blockDim.y) * 2;
    if (x + 1 >= nWidth || y + 1 >= nHeight) {
        return;
    }

    float3 shift_rgb{shift_r, shift_g, shift_b};

    uint8_t *pSrcY = pY + x * sizeof(YuvUnitx2) / 2 + y * yLinesize;
    uint8_t *pSrcUV = pUV + x * sizeof(YuvUnitx2) / 2 + y / 2 * uvLinesize;

    YuvUnitx2 l0 = *(YuvUnitx2 *)pSrcY;
    YuvUnitx2 l1 = *(YuvUnitx2 *)(pSrcY + yLinesize);
    YuvUnitx2 ch = *(YuvUnitx2 *)(pSrcUV);

    // if (x == 0 && y == 0) printf("Y[0], U[0], V[0] = %d, %d, %d\n", l0.x, ch.x, ch.y);
    
    Rgb rgb0 = normalizePixel(YuvToRgbForPixel<Rgb>(l0.x, ch.x, ch.y), norm, shift_rgb),
        rgb1 = normalizePixel(YuvToRgbForPixel<Rgb>(l0.y, ch.x, ch.y), norm, shift_rgb),
        rgb2 = normalizePixel(YuvToRgbForPixel<Rgb>(l1.x, ch.x, ch.y), norm, shift_rgb),
        rgb3 = normalizePixel(YuvToRgbForPixel<Rgb>(l1.y, ch.x, ch.y), norm, shift_rgb);

    uint8_t *pDst1 = p1 + x * sizeof(RgbUnitx2) / 2 + y * rgbpLinesize;
    uint8_t *pDst2 = p2 + x * sizeof(RgbUnitx2) / 2 + y * rgbpLinesize;
    uint8_t *pDst3 = p3 + x * sizeof(RgbUnitx2) / 2 + y * rgbpLinesize;
    *(RgbUnitx2 *)pDst1 = RgbUnitx2 {rgb0.v.x, rgb1.v.x};
    // if (x == 0 && y == 0) printf("R[0] = %f\n", rgb0.v.x);
    *(RgbUnitx2 *)(pDst1 + rgbpLinesize) = RgbUnitx2 {rgb2.v.x, rgb3.v.x};
    // if (x == 0 && y == 0) printf("R[%d] = %f\n", rgbpLinesize, rgb2.v.x);
    // pDst += nRgbpPitch * nHeight;
    *(RgbUnitx2 *)pDst2 = RgbUnitx2 {rgb0.v.y, rgb1.v.y};
    *(RgbUnitx2 *)(pDst2 + rgbpLinesize) = RgbUnitx2 {rgb2.v.y, rgb3.v.y};
    // pDst += nRgbpPitch * nHeight;
    *(RgbUnitx2 *)pDst3 = RgbUnitx2 {rgb0.v.z, rgb1.v.z};
    *(RgbUnitx2 *)(pDst3 + rgbpLinesize) = RgbUnitx2 {rgb2.v.z, rgb3.v.z};
}

template<class YuvUnitx2, class Rgb, class RgbUnitx2>
__global__ static void Yuv444ToRgbPlanarKernel(uint8_t *pYuv, int nYuvPitch, uint8_t *pRgbp, int nRgbpPitch, int nWidth, int nHeight) {
    int x = (threadIdx.x + blockIdx.x * blockDim.x) * 2;
    int y = (threadIdx.y + blockIdx.y * blockDim.y);
    if (x + 1 >= nWidth || y >= nHeight) {
        return;
    }

    uint8_t *pSrc = pYuv + x * sizeof(YuvUnitx2) / 2 + y * nYuvPitch;

    YuvUnitx2 l0 = *(YuvUnitx2 *)pSrc;
    YuvUnitx2 ch1 = *(YuvUnitx2 *)(pSrc + (nHeight * nYuvPitch));
    YuvUnitx2 ch2 = *(YuvUnitx2 *)(pSrc + (2 * nHeight * nYuvPitch));

    Rgb rgb0 = YuvToRgbForPixel<Rgb>(l0.x, ch1.x, ch2.x),
        rgb1 = YuvToRgbForPixel<Rgb>(l0.y, ch1.y, ch2.y);


    uint8_t *pDst = pRgbp + x * sizeof(RgbUnitx2) / 2 + y * nRgbpPitch;
    *(RgbUnitx2 *)pDst = RgbUnitx2{ rgb0.v.x, rgb1.v.x };

    pDst += nRgbpPitch * nHeight;
    *(RgbUnitx2 *)pDst = RgbUnitx2{ rgb0.v.y, rgb1.v.y };

    pDst += nRgbpPitch * nHeight;
    *(RgbUnitx2 *)pDst = RgbUnitx2{ rgb0.v.z, rgb1.v.z };
}

template <class COLOR32>
void Nv12ToColor32(uint8_t *dpNv12, int nNv12Pitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix) {
    SetMatYuv2Rgb(iMatrix);
    YuvToRgbKernel<uchar2, COLOR32, uint2>
        <<<dim3((nWidth + 63) / 32 / 2, (nHeight + 3) / 2 / 2), dim3(32, 2)>>>
        (dpNv12, nNv12Pitch, dpBgra, nBgraPitch, nWidth, nHeight);
}

template <class COLOR64>
void Nv12ToColor64(uint8_t *dpNv12, int nNv12Pitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix) {
    SetMatYuv2Rgb(iMatrix);
    YuvToRgbKernel<uchar2, COLOR64, ulonglong2>
        <<<dim3((nWidth + 63) / 32 / 2, (nHeight + 3) / 2 / 2), dim3(32, 2)>>>
        (dpNv12, nNv12Pitch, dpBgra, nBgraPitch, nWidth, nHeight);
}

template <class COLOR32>
void Nv12ToColor32(uint8_t **dpNv12, int nNv12Pitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix) {
    SetMatYuv2Rgb(iMatrix);
    YuvToRgbKernel<uchar2, COLOR32, uint2>
        <<<dim3((nWidth + 63) / 32 / 2, (nHeight + 3) / 2 / 2), dim3(32, 2)>>>
        (dpNv12[0], nNv12Pitch, dpBgra, nBgraPitch, nWidth, nHeight);
}

template <class COLOR32>
void YUV444ToColor32(uint8_t *dpYUV444, int nPitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix) {
    SetMatYuv2Rgb(iMatrix);
    Yuv444ToRgbKernel<uchar2, COLOR32, uint2>
        <<<dim3((nWidth + 63) / 32 / 2, (nHeight + 3) / 2), dim3(32, 2) >>>
        (dpYUV444, nPitch, dpBgra, nBgraPitch, nWidth, nHeight);
}

template <class COLOR64>
void YUV444ToColor64(uint8_t *dpYUV444, int nPitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix) {
    SetMatYuv2Rgb(iMatrix);
    Yuv444ToRgbKernel<uchar2, COLOR64, ulonglong2>
        <<<dim3((nWidth + 63) / 32 / 2, (nHeight + 3) / 2), dim3(32, 2) >>>
        (dpYUV444, nPitch, dpBgra, nBgraPitch, nWidth, nHeight);
}

template <class COLOR32>
void P016ToColor32(uint8_t *dpP016, int nP016Pitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix) {
    SetMatYuv2Rgb(iMatrix);
    YuvToRgbKernel<ushort2, COLOR32, uint2>
        <<<dim3((nWidth + 63) / 32 / 2, (nHeight + 3) / 2 / 2), dim3(32, 2)>>>
        (dpP016, nP016Pitch, dpBgra, nBgraPitch, nWidth, nHeight);
}

template <class COLOR64>
void P016ToColor64(uint8_t *dpP016, int nP016Pitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix) {
    SetMatYuv2Rgb(iMatrix);
    YuvToRgbKernel<ushort2, COLOR64, ulonglong2>
        <<<dim3((nWidth + 63) / 32 / 2, (nHeight + 3) / 2 / 2), dim3(32, 2)>>>
        (dpP016, nP016Pitch, dpBgra, nBgraPitch, nWidth, nHeight);
}

template <class COLOR32>
void YUV444P16ToColor32(uint8_t *dpYUV444, int nPitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix) {
    SetMatYuv2Rgb(iMatrix);
    Yuv444ToRgbKernel<ushort2, COLOR32, uint2>
        <<<dim3((nWidth + 63) / 32 / 2, (nHeight + 3) / 2), dim3(32, 2) >>>
        (dpYUV444, nPitch, dpBgra, nBgraPitch, nWidth, nHeight);
}

template <class COLOR64>
void YUV444P16ToColor64(uint8_t *dpYUV444, int nPitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix) {
    SetMatYuv2Rgb(iMatrix);
    Yuv444ToRgbKernel<ushort2, COLOR64, ulonglong2>
        <<<dim3((nWidth + 63) / 32 / 2, (nHeight + 3) / 2), dim3(32, 2) >>>
        (dpYUV444, nPitch, dpBgra, nBgraPitch, nWidth, nHeight);
}

template <class COLOR32>
void Nv12ToColorPlanar(uint8_t *dpNv12, int nNv12Pitch, uint8_t *dpBgrp, int nBgrpPitch, int nWidth, int nHeight, int iMatrix) {
    SetMatYuv2Rgb(iMatrix);
    YuvToRgbPlanarKernel<uchar2, COLOR32, uchar2>
        <<<dim3((nWidth + 63) / 32 / 2, (nHeight + 3) / 2 / 2), dim3(32, 2)>>>
        (dpNv12, nNv12Pitch, dpBgrp, nBgrpPitch, nWidth, nHeight);
}

template <class COLOR32>
void P016ToColorPlanar(uint8_t *dpP016, int nP016Pitch, uint8_t *dpBgrp, int nBgrpPitch, int nWidth, int nHeight, int iMatrix) {
    SetMatYuv2Rgb(iMatrix);
    YuvToRgbPlanarKernel<ushort2, COLOR32, uchar2>
        <<<dim3((nWidth + 63) / 32 / 2, (nHeight + 3) / 2 / 2), dim3(32, 2)>>>
        (dpP016, nP016Pitch, dpBgrp, nBgrpPitch, nWidth, nHeight);
}

template <class COLOR32>
void YUV444ToColorPlanar(uint8_t *dpYUV444, int nPitch, uint8_t *dpBgrp, int nBgrpPitch, int nWidth, int nHeight, int iMatrix) {
    SetMatYuv2Rgb(iMatrix);
    Yuv444ToRgbPlanarKernel<uchar2, COLOR32, uchar2>
        <<<dim3((nWidth + 63) / 32 / 2, (nHeight + 3) / 2), dim3(32, 2) >>>
        (dpYUV444, nPitch, dpBgrp, nBgrpPitch, nWidth, nHeight);
}

template <class COLOR32>
void YUV444P16ToColorPlanar(uint8_t *dpYUV444, int nPitch, uint8_t *dpBgrp, int nBgrpPitch, int nWidth, int nHeight, int iMatrix) {
    SetMatYuv2Rgb(iMatrix);
    Yuv444ToRgbPlanarKernel<ushort2, COLOR32, uchar2>
        << <dim3((nWidth + 63) / 32 / 2, (nHeight + 3) / 2), dim3(32, 2) >> >
        (dpYUV444, nPitch, dpBgrp, nBgrpPitch, nWidth, nHeight);
}

// Explicit Instantiation
template void Nv12ToColor32<BGRA32>(uint8_t *dpNv12, int nNv12Pitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix);
template void Nv12ToColor32<RGBA32>(uint8_t *dpNv12, int nNv12Pitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix);
template void Nv12ToColor32<RGBA32>(uint8_t **dpNv12, int nNv12Pitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix);
template void Nv12ToColor64<BGRA64>(uint8_t *dpNv12, int nNv12Pitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix);
template void Nv12ToColor64<RGBA64>(uint8_t *dpNv12, int nNv12Pitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix);
template void YUV444ToColor32<BGRA32>(uint8_t *dpYUV444, int nPitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix);
template void YUV444ToColor32<RGBA32>(uint8_t *dpYUV444, int nPitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix);
template void YUV444ToColor64<BGRA64>(uint8_t *dpYUV444, int nPitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix);
template void YUV444ToColor64<RGBA64>(uint8_t *dpYUV444, int nPitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix);
template void P016ToColor32<BGRA32>(uint8_t *dpP016, int nP016Pitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix);
template void P016ToColor32<RGBA32>(uint8_t *dpP016, int nP016Pitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix);
template void P016ToColor64<BGRA64>(uint8_t *dpP016, int nP016Pitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix);
template void P016ToColor64<RGBA64>(uint8_t *dpP016, int nP016Pitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix);
template void YUV444P16ToColor32<BGRA32>(uint8_t *dpYUV444, int nPitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix);
template void YUV444P16ToColor32<RGBA32>(uint8_t *dpYUV444, int nPitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix);
template void YUV444P16ToColor64<BGRA64>(uint8_t *dpYUV444, int nPitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix);
template void YUV444P16ToColor64<RGBA64>(uint8_t *dpYUV444, int nPitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix);
template void Nv12ToColorPlanar<BGRA32>(uint8_t *dpNv12, int nNv12Pitch, uint8_t *dpBgrp, int nBgrpPitch, int nWidth, int nHeight, int iMatrix);
template void Nv12ToColorPlanar<RGBA32>(uint8_t *dpNv12, int nNv12Pitch, uint8_t *dpBgrp, int nBgrpPitch, int nWidth, int nHeight, int iMatrix);
template void P016ToColorPlanar<BGRA32>(uint8_t *dpP016, int nP016Pitch, uint8_t *dpBgrp, int nBgrpPitch, int nWidth, int nHeight, int iMatrix);
template void P016ToColorPlanar<RGBA32>(uint8_t *dpP016, int nP016Pitch, uint8_t *dpBgrp, int nBgrpPitch, int nWidth, int nHeight, int iMatrix);
template void YUV444ToColorPlanar<BGRA32>(uint8_t *dpYUV444, int nPitch, uint8_t *dpBgrp, int nBgrpPitch, int nWidth, int nHeight, int iMatrix);
template void YUV444ToColorPlanar<RGBA32>(uint8_t *dpYUV444, int nPitch, uint8_t *dpBgrp, int nBgrpPitch, int nWidth, int nHeight, int iMatrix);
template void YUV444P16ToColorPlanar<BGRA32>(uint8_t *dpYUV444, int nPitch, uint8_t *dpBgrp, int nBgrpPitch, int nWidth, int nHeight, int iMatrix);
template void YUV444P16ToColorPlanar<RGBA32>(uint8_t *dpYUV444, int nPitch, uint8_t *dpBgrp, int nBgrpPitch, int nWidth, int nHeight, int iMatrix);

template<class YuvUnit, class RgbUnit>
__device__ inline YuvUnit RgbToY(RgbUnit r, RgbUnit g, RgbUnit b) {
    const YuvUnit low = 1 << (sizeof(YuvUnit) * 8 - 4);
    return matRgb2Yuv[0][0] * r + matRgb2Yuv[0][1] * g + matRgb2Yuv[0][2] * b + low;
}

template<class YuvUnit, class RgbUnit>
__device__ inline YuvUnit RgbToU(RgbUnit r, RgbUnit g, RgbUnit b) {
    const YuvUnit mid = 1 << (sizeof(YuvUnit) * 8 - 1);
    return matRgb2Yuv[1][0] * r + matRgb2Yuv[1][1] * g + matRgb2Yuv[1][2] * b + mid;
}

template<class YuvUnit, class RgbUnit>
__device__ inline YuvUnit RgbToV(RgbUnit r, RgbUnit g, RgbUnit b) {
    const YuvUnit mid = 1 << (sizeof(YuvUnit) * 8 - 1);
    return matRgb2Yuv[2][0] * r + matRgb2Yuv[2][1] * g + matRgb2Yuv[2][2] * b + mid;
}

template<class YuvUnitx2, class Rgb, class RgbIntx2>
__global__ static void RgbToYuvKernel(uint8_t *pRgb, int nRgbPitch, uint8_t *pYuv, int nYuvPitch, int nWidth, int nHeight) {
    int x = (threadIdx.x + blockIdx.x * blockDim.x) * 2;
    int y = (threadIdx.y + blockIdx.y * blockDim.y) * 2;
    if (x + 1 >= nWidth || y + 1 >= nHeight) {
        return;
    }

    uint8_t *pSrc = pRgb + x * sizeof(Rgb) + y * nRgbPitch;
    RgbIntx2 int2a = *(RgbIntx2 *)pSrc;
    RgbIntx2 int2b = *(RgbIntx2 *)(pSrc + nRgbPitch);

    Rgb rgb[4] = {int2a.x, int2a.y, int2b.x, int2b.y};
    decltype(Rgb::c.r)
        r = (rgb[0].c.r + rgb[1].c.r + rgb[2].c.r + rgb[3].c.r) / 4,
        g = (rgb[0].c.g + rgb[1].c.g + rgb[2].c.g + rgb[3].c.g) / 4,
        b = (rgb[0].c.b + rgb[1].c.b + rgb[2].c.b + rgb[3].c.b) / 4;

    uint8_t *pDst = pYuv + x * sizeof(YuvUnitx2) / 2 + y * nYuvPitch;
    *(YuvUnitx2 *)pDst = YuvUnitx2 {
        RgbToY<decltype(YuvUnitx2::x)>(rgb[0].c.r, rgb[0].c.g, rgb[0].c.b),
        RgbToY<decltype(YuvUnitx2::x)>(rgb[1].c.r, rgb[1].c.g, rgb[1].c.b),
    };
    *(YuvUnitx2 *)(pDst + nYuvPitch) = YuvUnitx2 {
        RgbToY<decltype(YuvUnitx2::x)>(rgb[2].c.r, rgb[2].c.g, rgb[2].c.b),
        RgbToY<decltype(YuvUnitx2::x)>(rgb[3].c.r, rgb[3].c.g, rgb[3].c.b),
    };
    *(YuvUnitx2 *)(pDst + (nHeight - y / 2) * nYuvPitch) = YuvUnitx2 {
        RgbToU<decltype(YuvUnitx2::x)>(r, g, b),
        RgbToV<decltype(YuvUnitx2::x)>(r, g, b),
    };
}


static __device__ float2 operator*(float2 x, float y) { return make_float2(x.x * y, x.y * y); }
template<class YuvUnitx2, class Rgb, class RgbIntx2>
__global__ static void RgbpToYuvKernel(uint8_t *pRgb, int nRgbPitch, uint8_t *pYuv, int nYuvPitch, int nWidth, int nHeight) {
    int x = (threadIdx.x + blockIdx.x * blockDim.x) * 2;
    int y = (threadIdx.y + blockIdx.y * blockDim.y) * 2;
    if (x + 1 >= nWidth || y + 1 >= nHeight) {
        return;
    }

    float max = (1 << sizeof(YuvUnitx2::x) * 8) - 1.0f;
    // if (x == 0 && y == 0) printf("max = %f\n", max);

    // uint8_t *pSrc = pRgb + x * sizeof(Rgb) + y * nRgbPitch;
    // RgbIntx2 int2a = *(RgbIntx2 *)pSrc;
    // RgbIntx2 int2b = *(RgbIntx2 *)(pSrc + nRgbPitch);

    uint8_t *pSrcR = pRgb + x * sizeof(Rgb::c.r) + y * nRgbPitch;
    uint8_t *pSrcG = pRgb + nHeight * nRgbPitch + x * sizeof(Rgb::c.r) + y * nRgbPitch;
    uint8_t *pSrcB = pRgb + nHeight * nRgbPitch * 2 + x * sizeof(Rgb::c.r) + y * nRgbPitch;

    RgbIntx2 int2aR = *(RgbIntx2 *)pSrcR * max, int2bR = *(RgbIntx2 *)(pSrcR + nRgbPitch) * max;
    RgbIntx2 int2aG = *(RgbIntx2 *)pSrcG * max, int2bG = *(RgbIntx2 *)(pSrcG + nRgbPitch) * max;
    RgbIntx2 int2aB = *(RgbIntx2 *)pSrcB * max, int2bB = *(RgbIntx2 *)(pSrcB + nRgbPitch) * max;

    // if (x == 0 && y == 0) printf("R[0], G[0], B[0] = %f, %f, %f\n", int2aR.x, int2aG.x, int2aB.x);

    // Rgb rgb[4] = {int2a.x, int2a.y, int2b.x, int2b.y};
    // decltype(Rgb::c.r)
    //     r = (rgb[0].c.r + rgb[1].c.r + rgb[2].c.r + rgb[3].c.r) / 4,
    //     g = (rgb[0].c.g + rgb[1].c.g + rgb[2].c.g + rgb[3].c.g) / 4,
    //     b = (rgb[0].c.b + rgb[1].c.b + rgb[2].c.b + rgb[3].c.b) / 4;

    decltype(Rgb::c.r)
        r = (int2aR.x + int2aR.y + int2bR.x + int2bR.y) / 4,
        g = (int2aG.x + int2aG.y + int2bG.x + int2bG.y) / 4,
        b = (int2aB.x + int2aB.y + int2bB.x + int2bB.y) / 4;

    uint8_t *pDst = pYuv + x * sizeof(YuvUnitx2) / 2 + y * nYuvPitch;
    *(YuvUnitx2 *)pDst = YuvUnitx2 {
        RgbToY<decltype(YuvUnitx2::x)>(int2aR.x, int2aG.x, int2aB.x),
        RgbToY<decltype(YuvUnitx2::x)>(int2aR.y, int2aG.y, int2aB.y),
    };
    *(YuvUnitx2 *)(pDst + nYuvPitch) = YuvUnitx2 {
        RgbToY<decltype(YuvUnitx2::x)>(int2bR.x, int2bG.x, int2bB.x),
        RgbToY<decltype(YuvUnitx2::x)>(int2bR.y, int2bG.y, int2bB.y),
    };
    *(YuvUnitx2 *)(pDst + (nHeight - y / 2) * nYuvPitch) = YuvUnitx2 {
        RgbToU<decltype(YuvUnitx2::x)>(r, g, b),
        RgbToV<decltype(YuvUnitx2::x)>(r, g, b),
    };
}

void Bgra64ToP016(uint8_t *dpBgra, int nBgraPitch, uint8_t *dpP016, int nP016Pitch, int nWidth, int nHeight, int iMatrix) {
    SetMatRgb2Yuv(iMatrix);
    RgbToYuvKernel<ushort2, BGRA64, ulonglong2>
        <<<dim3((nWidth + 63) / 32 / 2, (nHeight + 3) / 2 / 2), dim3(32, 2)>>>
        (dpBgra, nBgraPitch, dpP016, nP016Pitch, nWidth, nHeight);
}

template<class COLOR32>
void Color32ToNv12(uint8_t *dpBgra, int nBgraPitch, uint8_t *dpNv12, int nNv12Pitch, int nWidth, int nHeight, int iMatrix){
    SetMatRgb2Yuv(iMatrix);
    RgbToYuvKernel<uchar2, COLOR32, uint2>
            <<<dim3((nWidth + 63) / 32 / 2, (nHeight + 3) / 2 / 2), dim3(32, 2)>>>
            (dpBgra, nBgraPitch, dpNv12, nNv12Pitch, nWidth, nHeight);
}

template void Color32ToNv12<RGBA32>(uint8_t *dpBgra, int nBgraPitch, uint8_t *dpNv12, int nNv12Pitch, int nWidth, int nHeight, int iMatrix);

#ifdef __cplusplus
extern "C"
{
#endif
void nv12_to_rgbpf32(hipStream_t stream, uint8_t **dp_nv12, int *nv12_pitch, uint8_t **dp_rgbpf32, int *rgbpf32_pitch, int width, int height, int matrix)
{
    SetMatYuv2Rgb(matrix, stream);
    YuvToRgbPlanarKernel<uchar2, RGBAF32, float2>
    <<<dim3((width + 63) / 32 / 2, (height + 3) / 2 / 2), dim3(32, 2), 0, stream>>>
    (dp_nv12[0], dp_nv12[1], nv12_pitch[0], nv12_pitch[1], dp_rgbpf32[0], dp_rgbpf32[1],
        dp_rgbpf32[2], rgbpf32_pitch[0], width, height);
}
void nv12_to_rgbpf32_shift(hipStream_t stream, uint8_t **dp_nv12, int *nv12_pitch, uint8_t **dp_rgbpf32, int *rgbpf32_pitch,
    int width, int height, float norm, float* shift, int matrix)
{
    SetMatYuv2Rgb(matrix, stream);
    YuvToRgbPlanarKernel<uchar2, RGBAF32, float2>
    <<<dim3((width + 63) / 32 / 2, (height + 3) / 2 / 2), dim3(32, 2), 0, stream>>>
    (dp_nv12[0], dp_nv12[1], nv12_pitch[0], nv12_pitch[1], dp_rgbpf32[0], dp_rgbpf32[1],
        dp_rgbpf32[2], rgbpf32_pitch[0], width, height, norm, shift[0], shift[1], shift[2]);
}
void nv12_to_bgrpf32_shift(hipStream_t stream, uint8_t **dp_nv12, int *nv12_pitch, uint8_t **dp_rgbpf32, int *rgbpf32_pitch,
    int width, int height, float norm, float* shift, int matrix)
{
    SetMatYuv2Rgb(matrix, stream);
    YuvToRgbPlanarKernel<uchar2, BGRAF32, float2>
    <<<dim3((width + 63) / 32 / 2, (height + 3) / 2 / 2), dim3(32, 2), 0, stream>>>
    (dp_nv12[0], dp_nv12[1], nv12_pitch[0], nv12_pitch[1], dp_rgbpf32[0], dp_rgbpf32[1],
        dp_rgbpf32[2], rgbpf32_pitch[0], width, height, norm, shift[0], shift[1], shift[2]);
}
// void nv12_to_rgbpf32(hipStream_t stream, uint8_t **dp_nv12, int *nv12_pitch, uint8_t *dp_rgbpf32, int rgbpf32_pitch, int width, int height, int matrix)
// {
//     uint8_t* dp_rgbpf32_data[3];
//     dp_rgbpf32_data[0] = dp_rgbpf32;
//     dp_rgbpf32_data[1] = dp_rgbpf32 + rgbpf32_pitch * height;
//     dp_rgbpf32_data[2] = dp_rgbpf32 + rgbpf32_pitch * height * 2;

//     SetMatYuv2Rgb(matrix, stream);
//     YuvToRgbPlanarKernel<uchar2, RGBAF32, float2>
//     <<<dim3((width + 63) / 32 / 2, (height + 3) / 2 / 2), dim3(32, 2), 0, stream>>>
//     (dp_nv12[0], dp_nv12[1], nv12_pitch[0], nv12_pitch[1], dp_rgbpf32_data[0], dp_rgbpf32_data[1],
//         dp_rgbpf32_data[2], rgbpf32_pitch, width, height);
// }

// void nv12_to_rgbpf32chw(CUStream stream, uint8_t **dp_nv12, int *nv12_pitch, uint8_t **dp_rgbpf32, int *rgbpf32_pitch, int width, int height, int matrix)
void rgbpf32_to_nv12(hipStream_t stream, uint8_t **dp_rgbpf32, int *rgbpf32_pitch, uint8_t **dp_nv12, int *nv12_pitch, int width, int height, int matrix)
{
    SetMatRgb2Yuv(matrix, stream);
    RgbpToYuvKernel<uchar2, RGBAF32, float2>
        <<<dim3((width + 63) / 32 / 2, (height + 3) / 2 / 2), dim3(32, 2), 0 , stream>>>
        (dp_rgbpf32[0], rgbpf32_pitch[0], dp_nv12[0], nv12_pitch[0], width, height);
}
#ifdef __cplusplus
}
#endif